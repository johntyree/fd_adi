#include "hip/hip_runtime.h"

#include "GNUC_47_compat.h"

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/version.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include "tiled_range.h"
#include "strided_range.h"

#include <algorithm>
#include <cstdlib>

#include <iostream>
#include <vector>
#include <iterator>
#include <cassert>
#include <stdexcept>

#include <sys/select.h>

#include <hipsparse.h>

#include "_TriBandedOperatorGPU.cuh"

template <typename T, typename U>
int find_index(T haystack, U needle, int max) {
    FULLTRACE;
    int idx;
    for (idx = 0; idx < max; ++idx) {
        if (haystack[idx] == needle) break;
    }
    if (idx >= max) {
        /* LOG("Did not find "<<needle<<" before reaching index "<<max<<"."); */
        /* std::cout << '\t'; */
        /* std::cout << "Haystack [ "; */
        /* for (idx = 0; idx < max; ++idx) { */
            /* std::cout << haystack[idx] << " "; */
        /* } */
        /* std::cout << "]"; ENDL; */
        idx = -1;
    }
    FULLTRACE;
    return idx;
}

_TriBandedOperator::_TriBandedOperator(
        SizedArray<double> &data,
        SizedArray<double> &R,
        SizedArray<int> &offsets,
        SizedArray<double> &high_dirichlet,
        SizedArray<double> &low_dirichlet,
        SizedArray<double> &top_factors,
        SizedArray<double> &bottom_factors,
        unsigned int axis,
        Py_ssize_t operator_rows,
        Py_ssize_t blocks,
        bool has_high_dirichlet,
        bool has_low_dirichlet,
        bool top_is_folded,
        bool bottom_is_folded,
        bool has_residual
        ) :
    diags(data),
    R(R),
    offsets(offsets),
    high_dirichlet(high_dirichlet),
    low_dirichlet(low_dirichlet),
    top_factors(top_factors),
    bottom_factors(bottom_factors),
    axis(axis),
    main_diag(find_index(offsets.data, 0, offsets.size)),
    operator_rows(operator_rows),
    blocks(blocks),
    block_len(operator_rows / blocks),
    sup(diags.data.ptr()),
    mid(diags.data.ptr() + operator_rows),
    sub(diags.data.ptr() + 2*operator_rows),
    has_high_dirichlet(has_high_dirichlet),
    has_low_dirichlet(has_low_dirichlet),
    top_is_folded(top_is_folded),
    bottom_is_folded(bottom_is_folded),
    has_residual(has_residual),
    is_tridiagonal(offsets.size == 3 && main_diag != -1)
    {
        verify_diag_ptrs();
        status = hipsparseCreate(&handle);
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            DIE("CUSPARSE Library initialization failed.");
        }
    }

void _TriBandedOperator::verify_diag_ptrs() {
    FULLTRACE;
    if (sup.get() == 0 || mid.get() == 0 || sub.get() == 0) {
        DIE("Diag pointers aren't non-null");
    }
    if (main_diag == -1) {
        /* LOG("No main diag means not tridiagonal, hopefully."); */
        return;
    }
    if (offsets.size != 3) {
        /* LOG("Not tridiagonal. Skipping diag ptrs check."); */
        return;
    }
    int idx;
    /* LOG("main_diag("<<main_diag<<")"); */
    idx = diags.idx(main_diag-1, 0);
    if (*sup != diags.data[idx]
            || (sup.get() != (&diags.data[diags.idx(0,0)]).get())) {
        DIE("sup[0] = " << *sup << " <->  " << diags.get(0,0)
            << "\n\tsup = " << sup.get() << " <->  "
                << (&diags.data[diags.idx(0, 0)]).get());
    }
    if (*mid != diags.data[diags.idx(main_diag, 0)]
            || (mid.get() != (&diags.data[diags.idx(main_diag, 0)]).get())) {
        DIE("mid[0] = " << *mid << " !=  " << diags.get(main_diag,0)
            << "\n\tmid = " << mid.get() << " <->  "
                << (&diags.data[diags.idx(main_diag, 0)]).get());
    }
    if (*sub != diags.data[diags.idx(main_diag+1, 0)]
            || (sub.get() != (&diags.data[diags.idx(main_diag+1, 0)]).get())) {
        DIE("sub[0] = " << *sub << " !=  " << diags.get(main_diag+1,0)
            << "\n\tsub = " << sub.get() << " <->  "
                << (&diags.data[diags.idx(main_diag+1, 0)]).get());
    }
    FULLTRACE;
}


struct zipdot3 : thrust::binary_function<const Triple &, const Triple &, REAL_t> {
    __host__ __device__
    REAL_t operator()(const Triple &diags, const Triple &x) {
        using thrust::get;
        const REAL_t a = get<0>(diags);
        const REAL_t b = get<1>(diags);
        const REAL_t c = get<2>(diags);
        const REAL_t x0 = get<0>(x);
        const REAL_t x1 = get<1>(x);
        const REAL_t x2 = get<2>(x);
        return a*x0 + b*x1 + c*x2;
    }
};

SizedArray<double> *_TriBandedOperator::apply(SizedArray<double> &V) {
    FULLTRACE;
    using std::cout;
    using std::endl;
    const unsigned N = V.size;
    GPUVec<REAL_t> out(N);

    GPUVec<REAL_t> &in = V.data;

    if (!is_tridiagonal) {
        DIE("Can only apply tridiagonal operators when on the GPU.");
    }


    if (has_low_dirichlet) {
        /* print "Setting V[0,:] to", self.dirichlet[0] */
        // Some kind of thrust thing?
        /* V[...,0] = low_dirichlet[i] */
        thrust::copy(low_dirichlet.data.begin(),
                low_dirichlet.data.end(),
                in.begin());
    }
    if (has_high_dirichlet) {
        /* print "Setting V[0,:] to", self.dirichlet[0] */
        // Some kind of thrust thing?
        /* V[...,-1] = high_dirichlet[i] */
        thrust::copy(high_dirichlet.data.begin(),
                high_dirichlet.data.end(),
                in.end() - V.shape[1]);
    }

    if (axis == 0) {
        V.transpose(1);
    }



    GPUVec<REAL_t> a(sub, sub+N);
    GPUVec<REAL_t> b(mid, mid+N);
    GPUVec<REAL_t> c(sup, sup+N);

    out[0] = b[0]*in[0] + c[0]*in[1];
    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(a.begin()+1, b.begin()+1, c.begin()+1)),
        thrust::make_zip_iterator(thrust::make_tuple(a.end()-1, b.end()-1, c.end()-1)),
        thrust::make_zip_iterator(thrust::make_tuple(in.begin(), in.begin()+1, in.begin()+2)),
        out.begin()+1,
        zipdot3()
    );
    out[N-1] = a[N-1]*in[N-2] + b[N-1]*in[N-1];

    if (is_folded()) {
        fold_vector(out, true);
    }


    if (has_residual) {
        thrust::transform(out.begin(), out.end(),
                R.data.begin(),
                out.begin(),
                thrust::plus<double>());
    }

    // TODO: We can transpose `out` straight into U
    SizedArray<double> *U = new SizedArray<double>(out,
            V.ndim, V.shape, "CPP Solve U from V");

    if (axis == 0) {
        U->transpose(1);
    }
    FULLTRACE;
    return U;
}


struct periodic_from_to_mask : thrust::unary_function<int, bool> {
    int begin;
    int end;
    int period;

    periodic_from_to_mask(int begin, int end, int period)
        : begin(begin-1), end(end+1), period(period) {
        }

    __host__ __device__
    bool operator()(int idx) {
        return (idx % period != begin && idx % period != end);
    }
};

void _TriBandedOperator::add_operator(_TriBandedOperator &other) {
    /* LOG("Adding operator @ " << &other << " to this one @ " << this); */
    /*
    * Add a second BandedOperator to this one.
    * Does not alter self.R, the residual vector.
    */
    FULLTRACE;
    int begin = has_low_dirichlet;
    int end = block_len-1 - has_high_dirichlet;
    int o, to, fro;
    for (int i = 0; i < other.offsets.size; i++) {
        fro = i;
        o = other.offsets.get(i);
        to = find_index(offsets.data, o, offsets.size);
        if (offsets.get(to) != o) {
            std::cout << std::endl;
            std::cout << "to: " << to << "(";
            /* print_array(&offsets(0), offsets.size); */
            std::cout << offsets.data;
            std::cout << ")";
            std::cout << "fro: " << fro << "(";
            std::cout << other.offsets.data;
            /* print_array(&other.offsets(0), other.offsets.size); */
            std::cout << ")" << std::endl;
            assert(offsets.get(to) == o);
        }
        /* LOG("Adding offset " << o << "."); */
        if (o == 0) {
            thrust::transform_if(
                    &diags.data[diags.idx(to, 0)],
                    &diags.data[diags.idx(to, 0)] + operator_rows,
                    &other.diags.data[diags.idx(fro, 0)],
                    thrust::make_counting_iterator(0),
                    &diags.data[diags.idx(to, 0)],
                    thrust::plus<double>(),
                    periodic_from_to_mask(begin, end, block_len));
        } else {
            thrust::transform(
                    &other.diags.data[diags.idx(fro, 0)],
                    &other.diags.data[diags.idx(fro, 0)] + other.diags.shape[1],
                    &diags.data[diags.idx(to, 0)],
                    &diags.data[diags.idx(to, 0)],
                    thrust::plus<double>());
        }
    }
    /* LOG("Adding R."); */
    thrust::transform(
            R.data.begin(),
            R.data.end(),
            other.R.data.begin(),
            R.data.begin(),
            thrust::plus<double>());
    FULLTRACE;
}



void _TriBandedOperator::add_scalar(double val) {
    FULLTRACE;
    /* Add a scalar to the main diagonal.
     * Does not alter the residual vector.
     */
    // We add it to the main diagonal.

    int begin = has_low_dirichlet;
    int end = block_len-1 - has_high_dirichlet;

    /* LOG("has_low("<<has_low_dirichlet<<") " */
        /* "has_high("<<has_high_dirichlet<<") " */
        /* "blocklen("<<block_len<<") "); */

    thrust::transform_if(
            &diags.data[diags.idx(main_diag, 0)],
            &diags.data[diags.idx(main_diag, 0)] + operator_rows,
            thrust::make_constant_iterator(val),
            thrust::make_counting_iterator(0),
            &diags.data[diags.idx(main_diag, 0)],
            thrust::plus<double>(),
            periodic_from_to_mask(begin, end, block_len));
    FULLTRACE;
}

bool _TriBandedOperator::is_folded() {
    return top_is_folded || bottom_is_folded;
}



int _TriBandedOperator::solve(SizedArray<double> &V) {
    FULLTRACE;
    if (!is_tridiagonal) {
        DIE("Can only solve tridiagonal systems when on the GPU.");
    }
    verify_diag_ptrs();

    /* std::cout << "Begin C Solve\n"; */
    /* std::cout << "Copy Host->Dev... " << V.data << ' '; */
    GPUVec<double> d_V(V.data);
    GPUVec<double> d_sup(sup, sup+V.size);
    GPUVec<double> d_mid(mid, mid+V.size);
    GPUVec<double> d_sub(sub, sub+V.size);
    /* std::cout << "OK\n"; */

    /* std::cout << "CUSPARSE... "; */
    status = cusparseDgtsvStridedBatch(handle, V.size,
            d_sub.raw(), d_mid.raw(), d_sup.raw(),
            d_V.raw(),
            1, V.size);
    hipDeviceSynchronize();
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << "CUSPARSE tridiag system solve failed." << std::endl;
        return 1;
    }
    /* std::cout << "OK\n"; */

    /* std::cout << "Copy Dev->Host... " << d_V << ' '; */
    thrust::copy(d_V.begin(), d_V.end(), V.data.begin());
    /* std::cout << "OK\n"; */
    /* std::cout << "End C Solve\n"; */
    FULLTRACE;
    return 0;
}

template <typename Tuple, typename OP>
struct curry : public thrust::unary_function<Tuple, typename OP::result_type> {

    OP f;

    __host__ __device__
    typename OP::result_type operator()(Tuple t) {
        using thrust::get;
        return  f(get<0>(t), get<1>(t));
    }
};

template <typename Tuple, typename Result>
struct add_multiply3 : public thrust::unary_function<Tuple, Result> {
    Result direction;
    add_multiply3(Result x) : direction(x) {}
    __host__ __device__
    Result operator()(Tuple t) {
        using thrust::get;
        return  get<0>(t) + direction * get<1>(t) * get<2>(t);
    }
};


void _TriBandedOperator::fold_vector(GPUVec<double> &vector, bool unfold) {
    FULLTRACE;

    using thrust::make_zip_iterator;
    using thrust::make_tuple;

    typedef GPUVec<REAL_t>::iterator Iterator;
    typedef thrust::tuple<REAL_t,REAL_t,REAL_t> REALTuple;

    strided_range<Iterator> u0(vector.begin(), vector.end(), block_len);
    strided_range<Iterator> u1(vector.begin()+1, vector.end(), block_len);

    strided_range<Iterator> un(vector.begin()+block_len-1, vector.end(), block_len);
    strided_range<Iterator> un1(vector.begin()+block_len-2, vector.end(), block_len);

    LOG("top_is_folded("<<top_is_folded<<") bottom_is_folded("<<bottom_is_folded<<")");
    // Top fold
    if (top_is_folded) {
        /* LOG("Folding top. direction("<<unfold<<") top_factors("<<top_factors<<")"); */
        thrust::transform(
            make_zip_iterator(make_tuple(u0.begin(), u1.begin(), top_factors.data.begin())),
            make_zip_iterator(make_tuple(u0.end(), u1.end(), top_factors.data.end())),
            u0.begin(),
            add_multiply3<REALTuple, REAL_t>(unfold ? -1 : 1));
    }

    if (bottom_is_folded) {
        /* LOG("Folding bottom. direction("<<unfold<<") bottom_factors("<<bottom_factors<<")"); */
        thrust::transform(
            make_zip_iterator(make_tuple(un.begin(), un1.begin(), bottom_factors.data.begin())),
            make_zip_iterator(make_tuple(un.end(), un1.end(), bottom_factors.data.end())),
            un.begin(),
            add_multiply3<REALTuple, REAL_t>(unfold ? -1 : 1));
    }

    FULLTRACE;
}


void _TriBandedOperator::diagonalize() {
    FULLTRACE;
    if (bottom_is_folded) fold_bottom();
    if (top_is_folded) fold_top();
    FULLTRACE;
}

void _TriBandedOperator::undiagonalize() {
    FULLTRACE;
    if (bottom_is_folded) fold_bottom(true);
    if (top_is_folded) fold_top(true);
    FULLTRACE;
}


template <typename Tuple>
struct fold_operator : public thrust::unary_function<Tuple, void> {
    bool unfold;
    fold_operator(bool x) : unfold(x) {}
    __host__ __device__
    void operator()(Tuple t) {
        using thrust::get;
        int const c0   = 0;
        int const c1   = 1;
        int const b0   = 2;
        int const b1   = 3;
        int const a1   = 4;
        int const fact = 5;
        int nothing = c0 + c1 + b0 + b1 + a1 + fact;
        nothing = nothing;
        if (unfold) {
            get<c0>(t) -= get<b1>(t) * get<fact>(t);
            get<b0>(t) -= get<a1>(t) * get<fact>(t);
            get<fact>(t) *= -get<c1>(t);
        } else {
            get<fact>(t) = get<c1>(t) == 0 ? 0 : -get<fact>(t) / get<c1>(t);
            get<c0>(t) += get<b1>(t) * get<fact>(t);
            get<b0>(t) += get<a1>(t) * get<fact>(t);
        }
    }
};

void _TriBandedOperator::fold_top(bool unfold) {
    FULLTRACE;
    typedef thrust::tuple<REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&> REALTuple;
    typedef thrust::device_ptr<REAL_t> Ptr;

    strided_range<Ptr> c0 (sup  , sup+operator_rows, block_len);
    strided_range<Ptr> c1 (sup+1, sup+operator_rows, block_len);
    strided_range<Ptr> b0 (mid  , mid+operator_rows, block_len);
    strided_range<Ptr> b1 (mid+1, mid+operator_rows, block_len);
    strided_range<Ptr> a1 (sub+1, sub+operator_rows, block_len);

    thrust::for_each(
        make_zip_iterator(
            make_tuple(
                c0.begin(), c1.begin(),
                b0.begin(), b1.begin(),
                            a1.begin(),
                top_factors.data.begin()
            )
        ),
        make_zip_iterator(
            make_tuple(
                c0.end(), c1.end(),
                b0.end(), b1.end(),
                          a1.end(),
                top_factors.data.end()
            )
        ),
        fold_operator<REALTuple>(unfold)
    );

    top_is_folded = !unfold;
    FULLTRACE;
}


void _TriBandedOperator::fold_bottom(bool unfold) {
    FULLTRACE;
    typedef thrust::tuple<REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&> REALTuple;
    typedef thrust::device_ptr<REAL_t> Ptr;

    strided_range<Ptr> cn1(sup+(block_len-2)  , sup+operator_rows, block_len);
    strided_range<Ptr> bn (mid+(block_len-1)  , mid+operator_rows, block_len);
    strided_range<Ptr> bn1(mid+(block_len-1)-1, mid+operator_rows, block_len);
    strided_range<Ptr> an (sub+(block_len-1), sub+operator_rows, block_len);
    strided_range<Ptr> an1(sub+(block_len-1)-1, sub+operator_rows, block_len);

    thrust::for_each(
        make_zip_iterator(
            make_tuple(
                an.begin(), an1.begin(),
                bn.begin(), bn1.begin(),
                            cn1.begin(),
                bottom_factors.data.begin()
            )
        ),
        make_zip_iterator(
            make_tuple(
                an.end(), an1.end(),
                bn.end(), bn1.end(),
                          cn1.end(),
                bottom_factors.data.end()
            )
        ),
        fold_operator<REALTuple>(unfold)
    );

    bottom_is_folded = !unfold;
    FULLTRACE;
}


void _TriBandedOperator::vectorized_scale(SizedArray<double> &vector) {
    FULLTRACE;
    Py_ssize_t vsize = vector.size;
    Py_ssize_t noffsets = offsets.size;
    Py_ssize_t block_len = operator_rows / blocks;

    typedef thrust::device_vector<REAL_t>::iterator Iterator;
    tiled_range<Iterator> v(vector.data.begin(), vector.data.end(), operator_rows / vsize);
    /*
     * LOG("op_rows("<<operator_rows<<") vsize("<<vsize<<") "
     *     "v.d.size("<<vector.data.size()<<") "
     *     "v.size()("<<v.end()-v.begin()<<") "
     *     "diags.shape("<<diags.shape[0]<<","<<diags.shape[1]<<") "
     *     "diags.idx(1,0)("<<diags.idx(1,0)<<") "
     *     );
     * LOG("diags.name("<<diags.name<<")");
     * LOG("diags.idx(0,op)("<<diags.idx(0,0)+operator_rows<<")");
     */

    if (operator_rows % vsize != 0) {
        DIE("Vector length does not divide "
            "evenly into operator size. Cannot scale."
            << "\n vsize("<<vsize<<") operator_rows("<<operator_rows<<")");
    }
    if ((size_t)vsize != vector.data.size()) {DIE("vsize != vector.data.size()")}
    if (vsize == 0) {DIE("vsize == 0")}

    if (has_low_dirichlet) {
        for (Py_ssize_t b = 0; b < blocks; ++b) {
            vector.data[vector.idx(b*block_len % vsize)] = 1;
        }
    }

    if (has_high_dirichlet) {
        for (Py_ssize_t b = 0; b < blocks; ++b) {
            vector.data[vector.idx((b+1)*block_len - 1 % vsize)] = 1;
        }
    }

    for (Py_ssize_t row = 0; row < noffsets; ++row) {
        int o = offsets.get(row);
        if (o >= 0) { // upper diags
            thrust::transform(diags.data.begin() + diags.idx(row, 0),
                    diags.data.begin() + diags.idx(row, 0) + operator_rows - o,
                    v.begin(),
                    diags.data.begin() + diags.idx(row, 0),
                    thrust::multiplies<REAL_t>());
        } else { // lower diags
            thrust::transform(diags.data.begin() + diags.idx(row, -o),
                    diags.data.begin() + diags.idx(row, 0) + operator_rows,
                    v.begin() + -o,
                    diags.data.begin() + diags.idx(row, -o),
                    thrust::multiplies<REAL_t>());
        }
    }
    /* LOG("Scaled data."); */
    thrust::transform(R.data.begin(), R.data.end(),
            v.begin(),
            R.data.begin(),
            thrust::multiplies<REAL_t>());
    /* LOG("Scaled R."); */
    FULLTRACE;
    return;
}

int main () {

    thrust::host_vector<double> a(10);
    int block_len = 5;
    int begin = 1;
    int end = block_len-1 - 1;

    thrust::transform_if(
            a.begin(),
            a.end(),
            thrust::make_constant_iterator(2),
            thrust::make_counting_iterator(0),
            a.begin(),
            thrust::plus<double>(),
            periodic_from_to_mask(begin, end, block_len));

    printf("\n");
    print_array(a.data(), a.size());
    return 0;
}
