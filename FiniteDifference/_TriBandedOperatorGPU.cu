#include "hip/hip_runtime.h"
#include "GNUC_47_compat.h"

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <stdexcept>
#include <vector>

#include <sys/select.h>

#include <hipsparse.h>

#include <thrust/adjacent_difference.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/detail/normal_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>
#include <thrust/version.h>

#include "tiled_range.h"
#include "strided_range.h"

#include "_TriBandedOperatorGPU.cuh"

using thrust::make_constant_iterator;
using thrust::make_counting_iterator;
using thrust::make_tuple;
using thrust::make_zip_iterator;

using namespace thrust::placeholders;

typedef thrust::device_ptr<double> Dptr;

typedef thrust::detail::normal_iterator<thrust::device_ptr<double> > DptrIterator;

const double NaN = std::numeric_limits<double>::quiet_NaN();


template <typename T, typename U>
int find_index(T haystack, U needle, int max) {
    FULLTRACE;
    int idx;
    for (idx = 0; idx < max; ++idx) {
        if (haystack[idx] == needle) break;
    }
    if (idx >= max) {
        /* LOG("Did not find "<<needle<<" before reaching index "<<max<<"."); */
        /* std::cout << '\t'; */
        /* std::cout << "Haystack [ "; */
        /* for (idx = 0; idx < max; ++idx) { */
            /* std::cout << haystack[idx] << " "; */
        /* } */
        /* std::cout << "]"; ENDL; */
        idx = -1;
    }
    FULLTRACE;
    return idx;
}

struct first_deriv {
    template <typename Tuple>
    __host__ __device__
    /* (sup, mid, sub, deltas+1, deltas+2) */
    void operator()(Tuple t) {
        using thrust::get;
        get<0>(t) = get<3>(t)                / (get<4>(t) * (get<3>(t) + get<4>(t)));
        get<1>(t) = (-get<3>(t) + get<4>(t)) / (get<3>(t) * get<4>(t));
        get<2>(t) = -get<4>(t)               / (get<3>(t) * (get<3>(t) + get<4>(t)));
    }
};

struct second_deriv {
    template <typename Tuple>
    __host__ __device__
    /* (sup, mid, sub, deltas+1, deltas+2) */
    void operator()(Tuple t) {
        using thrust::get;
        const double x = get<3>(t) + get<4>(t);
        get<0>(t) =  2. / (get<4>(t) * x);
        get<1>(t) = -2. / (get<4>(t)*get<3>(t));
        get<2>(t) =  2. / (get<3>(t) * x);
        /* get<0>(t) =  2.; */
        /* get<1>(t) = -2.; */
        /* get<2>(t) =  2.; */
    }
};

struct dirichlet_boundary {
    dirichlet_boundary(double val) : val(val) {}
    double val;
    template <typename Tuple>
    __host__ __device__
    // (sup, mid, sub, dirichlet)
    void operator()(Tuple t) {
        using thrust::get;
        get<0>(t) =  0;
        get<1>(t) =  1;
        get<2>(t) =  0;
        get<3>(t) = val;
    }
    /* # Dirichlet boundary. No derivatives, but we need to preserve the */
    /* # value we get, because we will have already forced it. */
    /* Bdata[m, 0] = 1 */
    /* B.dirichlet[0] = lower_val */
};

struct von_neumann_boundary {
// thrust::fill...
    von_neumann_boundary(double val) : val(val) {}

    double val;

    template <typename Tuple>
    __host__ __device__
    // (sup, mid, sub, residual)
    void operator()(Tuple t) {
        using thrust::get;
        get<0>(t) = 0;
        get<1>(t) = 0;
        get<2>(t) = 0;
        get<3>(t) = val;
    }
};

struct free_boundary_first {
    template <typename Tuple>
    __host__ __device__
    // (sup, mid, sub, d[1])
    // (mid, sub, sup, d[-1])
    void operator()(Tuple t) {
        using thrust::get;
        get<0>(t) =  1 / get<3>(t);
        get<1>(t) = -1 / get<3>(t);
        get<2>(t) = 0;
    }
    /* # Try first order to preserve tri-diag */
    /* Bdata[m - 1, 1] =  1 / d[1] */
    /* Bdata[m,     0] = -1 / d[1] */
    /* # First order backward */
    /* Bdata[m, -1]     =  1.0 / d[-1] */
    /* Bdata[m + 1, -2] = -1.0 / d[-1] */
};


struct free_boundary_second_with_first_derivative_one {
    template <typename Tuple>
    __host__ __device__
    // (sup, mid, sub, d[1], R[0])
    // (sub, mid, sup, -d[-1], R[-1]) yes mid is still neg
    void operator()(Tuple t) {
        using thrust::get;
        const double x = get<3>(t)*get<3>(t);
        const double fst_deriv = 1;
        get<0>(t) =  2 / x;
        get<1>(t) = -2 / x;
        get<2>(t) = 0;
        get<4>(t) = -fst_deriv*2 / get<3>(t);
    }
    /* Bdata[m-1, 1] =  2 / d[1]**2 */
    /* Bdata[m,   0] = -2 / d[1]**2 */
    /* R[0]         =  -fst_deriv * 2 / d[1] */
    /* Bdata[m,   -1] = -2 / d[-1]**2 */
    /* Bdata[m+1, -2] =  2 / d[-1]**2 */
    /* R[-1]          =  fst_deriv * 2 / d[-1] */
};

struct free_boundary_second {
    template <typename Tuple>
    __host__ __device__
    // (supsup, sup, mid, sub, d[1], d[2])
    // (subsub, sub, mid, sup, d[-1], d[-2])
    void operator()(Tuple t) {
        using thrust::get;
        const double recip_denom =
            1.0 / (0.5 * (get<4>(t)+get<5>(t))*get<4>(t)*get<5>(t));
        get<0>(t) = get<4>(t)              * recip_denom;
        get<1>(t) = -(get<4>(t)+get<5>(t)) * recip_denom;
        get<2>(t) = get<5>(t)              * recip_denom;
        get<3>(t) = 0;
    }
    /* recip_denom = 1.0 / (0.5*(d[2]+d[1])*d[2]*d[1]); */
    /* Bdata[m-2,2] = d[1]         * recip_denom */
    /* Bdata[m-1,1] = -(d[2]+d[1]) * recip_denom */
    /* Bdata[m,0]   = d[2]         * recip_denom */
    /* recip_denom = 1.0 / (0.5*(d[-2]+d[-1])*d[-2]*d[-1]); */
    /* Bdata[m+2,-3] = d[-1]          * recip_denom */
    /* Bdata[m+1,-2] = -(d[-2]+d[-1]) * recip_denom */
    /* Bdata[m,-1]   = d[-2]          * recip_denom */
};



template <typename T>
int spot_first(Dptr &sup, Dptr &mid, Dptr &sub, T deltas,
        Dptr &low_dirichlet, Dptr &residual, int sz, int blksz) {
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(sup+1, mid+1, sub+1, deltas+1, deltas+2)),
            thrust::make_zip_iterator(thrust::make_tuple(sup+sz-1, mid+sz-1, sub+sz-1, deltas+sz-1, deltas+sz)),
            first_deriv()
            );
    strided_range<DptrIterator> topsup(sup, sup+sz, blksz);
    strided_range<DptrIterator> topmid(mid, mid+sz, blksz);
    strided_range<DptrIterator> topsub(sub, sub+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(topsup.begin(), topmid.begin(), topsub.begin(), low_dirichlet)),
            thrust::make_zip_iterator(thrust::make_tuple(topsup.end(), topmid.end(), topsub.end(), low_dirichlet+1)),
            dirichlet_boundary(0)
            );
    strided_range<DptrIterator> botsup(sup+blksz-1, sup+sz, blksz);
    strided_range<DptrIterator> botmid(mid+blksz-1, mid+sz, blksz);
    strided_range<DptrIterator> botsub(sub+blksz-1, sub+sz, blksz);
    strided_range<DptrIterator> botresidual(residual+blksz-1, residual+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(botsup.begin(),
                    botmid.begin(), botsub.begin(), botresidual.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(botsup.end(),
                    botmid.end(), botsub.end(), botresidual.end())),
            von_neumann_boundary(1)
            );
    return 0;
}

template <typename T>
int spot_second(Dptr &sup, Dptr &mid, Dptr &sub, T deltas,
        Dptr &low_dirichlet, Dptr &residual, int sz, int blksz) {
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(sup+1, mid+1, sub+1, deltas+1, deltas+2)),
            thrust::make_zip_iterator(thrust::make_tuple(sup+sz-1, mid+sz-1, sub+sz-1, deltas+sz-1, deltas+sz)),
            second_deriv()
            );
    strided_range<DptrIterator> topsup(sup, sup+sz, blksz);
    strided_range<DptrIterator> topmid(mid, mid+sz, blksz);
    strided_range<DptrIterator> topsub(sub, sub+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(topsup.begin(), topmid.begin(), topsub.begin(), low_dirichlet)),
            thrust::make_zip_iterator(thrust::make_tuple(topsup.end(), topmid.end(), topsub.end(), low_dirichlet+1)),
            dirichlet_boundary(0)
            );
    strided_range<DptrIterator> botsup(sup+blksz-1, sup+sz, blksz);
    strided_range<DptrIterator> botmid(mid+blksz-1, mid+sz, blksz);
    strided_range<DptrIterator> botsub(sub+blksz-1, sub+sz, blksz);
    strided_range<thrust::transform_iterator<thrust::negate<double>, T> > botdel(
            make_transform_iterator(deltas+blksz-1, thrust::negate<double>()),
            make_transform_iterator(deltas+sz, thrust::negate<double>()),
            blksz);
    strided_range<DptrIterator> botresidual(residual+blksz-1, residual+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(
                    botsub.begin(), botmid.begin(), botsup.begin(),
                    botdel.begin(), botresidual.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(
                    botsub.end(), botmid.end(), botsup.end(),
                    botdel.end(), botresidual.end())),
            free_boundary_second_with_first_derivative_one()
            );
    return 0;
}

template <typename T>
int var_first(Dptr &sup, Dptr &mid, Dptr &sub, T deltas, int sz, int blksz) {
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(sup+1, mid+1, sub+1, deltas+1, deltas+2)),
            thrust::make_zip_iterator(thrust::make_tuple(sup+sz-1, mid+sz-1, sub+sz-1, deltas+sz-1, deltas+sz)),
            first_deriv()
            );
    strided_range<DptrIterator> topsup(sup, sup+sz, blksz);
    strided_range<DptrIterator> topmid(mid, mid+sz, blksz);
    strided_range<DptrIterator> topsub(sub, sub+sz, blksz);
    strided_range<T> topdel(deltas+1, deltas+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(topsup.begin(), topmid.begin(), topsub.begin(), topdel.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(topsup.end(), topmid.end(), topsub.end(), topdel.end())),
            free_boundary_first()
            );
    strided_range<DptrIterator> botsup(sup+blksz-1, sup+sz, blksz);
    strided_range<DptrIterator> botmid(mid+blksz-1, mid+sz, blksz);
    strided_range<DptrIterator> botsub(sub+blksz-1, sub+sz, blksz);
    strided_range<T> botdel(deltas+blksz-1, deltas+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(botmid.begin(), botsub.begin(), botsup.begin(), botdel.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(botmid.end(), botsub.end(), botsup.end(), botdel.end())),
            free_boundary_first()
            );
    return 0;
}

template <typename T>
int var_second(Dptr &sup, Dptr &mid, Dptr &sub, T deltas,
        Dptr &residual, Dptr &bottom_factors,
        int sz, int blksz) {
    int blks = sz / blksz;
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(sup+1, mid+1, sub+1, deltas+1, deltas+2)),
            thrust::make_zip_iterator(thrust::make_tuple(sup+sz-1, mid+sz-1, sub+sz-1, deltas+sz-1, deltas+sz)),
            second_deriv()
            );
    strided_range<DptrIterator> topsup(sup, sup+sz, blksz);
    strided_range<DptrIterator> topmid(mid, mid+sz, blksz);
    strided_range<DptrIterator> topsub(sub, sub+sz, blksz);
    strided_range<DptrIterator> topresidual(residual, residual+sz, blksz);
    /* We use the von neumann boundary here because we know this will all be
     * multiplied by the 0 spot value. */
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(topsup.begin(),
                    topmid.begin(), topsub.begin(), topresidual.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(topsup.end(),
                    topmid.end(), topsub.end(), topresidual.end())),
            von_neumann_boundary(0)
            );
    /* thrust::for_each( */
            /* thrust::make_zip_iterator(thrust::make_tuple(top_factors, */
                    /* topsup.begin(), topmid.begin(), topsub.begin(), */
                    /* topdel.begin(), topdel2.begin())), */
            /* thrust::make_zip_iterator(thrust::make_tuple(top_factors + blks, */
                    /* topsup.end(), topmid.end(), topsub.end(), */
                    /* topdel.end(), topdel2.end())), */
            /* free_boundary_second() */
            /* ); */
    strided_range<DptrIterator> botsup(sup+blksz-1, sup+sz, blksz);
    strided_range<DptrIterator> botmid(mid+blksz-1, mid+sz, blksz);
    strided_range<DptrIterator> botsub(sub+blksz-1, sub+sz, blksz);
    strided_range<T> botdel(deltas+blksz-1, deltas+sz, blksz);
    strided_range<T> botdel2(deltas+blksz-2, deltas+sz, blksz);
    thrust::for_each(
            thrust::make_zip_iterator(thrust::make_tuple(bottom_factors,
                    botsub.begin(), botmid.begin(), botsup.begin(),
                    botdel.begin(), botdel2.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(bottom_factors + blks,
                    botsub.end(), botmid.end(), botsup.end(),
                    botdel.end(), botdel2.end())),
            free_boundary_second()
            );
    return 0;
}

_TriBandedOperator::_TriBandedOperator(
        SizedArray<double> &data,
        SizedArray<double> &R,
        SizedArray<double> &high_dirichlet,
        SizedArray<double> &low_dirichlet,
        SizedArray<double> &top_factors,
        SizedArray<double> &bottom_factors,
        unsigned int axis,
        Py_ssize_t operator_rows,
        Py_ssize_t blocks,
        bool has_high_dirichlet,
        bool has_low_dirichlet,
        std::string top_fold_status,
        std::string bottom_fold_status,
        bool has_residual
        ) :
    diags(data, true),
    R(R, true),
    high_dirichlet(high_dirichlet, true),
    low_dirichlet(low_dirichlet, true),
    top_factors(top_factors, true),
    bottom_factors(bottom_factors, true),
    axis(axis),
    main_diag(1),
    operator_rows(operator_rows),
    blocks(blocks),
    block_len(operator_rows / blocks),
    sup(diags.data),
    mid(diags.data + operator_rows),
    sub(diags.data + 2*operator_rows),
    has_high_dirichlet(has_high_dirichlet),
    has_low_dirichlet(has_low_dirichlet),
    top_fold_status(top_fold_status),
    bottom_fold_status(bottom_fold_status),
    has_residual(has_residual)
    {
        /* LOG("TriBandedOperator initialize START."); */
        verify_diag_ptrs();
        status = hipsparseCreate(&handle);
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            DIE("CUSPARSE Library initialization failed.");
        }
        /* LOG("TriBandedOperator initialize END."); */
    }


void _TriBandedOperator::verify_diag_ptrs() {
    FULLTRACE;

    if (sup.get() == 0 || mid.get() == 0 || sub.get() == 0) {
        DIE("Diag pointers aren't non-null");
    }

    FULLTRACE;
}

struct zipdot3 {
    template <typename T>
    __host__ __device__
    REAL_t operator()(const T &diags, const T &x) {
        using thrust::get;
        const REAL_t a = get<0>(diags);
        const REAL_t b = get<1>(diags);
        const REAL_t c = get<2>(diags);
        const REAL_t x0 = get<0>(x);
        const REAL_t x1 = get<1>(x);
        const REAL_t x2 = get<2>(x);
        return a*x0 + b*x1 + c*x2;
    }
};
struct zipdotTopAndBottom {
    template <typename T>
    __host__ __device__
    void operator()(T t) {
        using thrust::get;

        REAL_t &ret = get<0>(t);
        const REAL_t &a  = get<1>(t);
        const REAL_t &x0 = get<2>(t);
        const REAL_t &b  = get<3>(t);
        const REAL_t &x1 = get<4>(t);

        ret = a*x0 + b*x1;

        REAL_t &retN = get<5>(t);
        const REAL_t &b1  = get<6>(t);
        const REAL_t &xN1 = get<7>(t);
        const REAL_t &c1  = get<8>(t);
        const REAL_t &xN  = get<9>(t);

        retN = b1*xN1 + c1*xN;
    }
};


void _TriBandedOperator::apply(SizedArray<double> &V) {
    FULLTRACE;
    if (top_fold_status == CAN_FOLD || bottom_fold_status == CAN_FOLD) {
        DIE("Must be tridiagonal to apply operator on GPU.");
    }
    const unsigned N = V.size;

    // TODO: This Iterator works, but is it right?
    strided_range<DptrIterator> u0(V.data, V.data+V.size, block_len);
    strided_range<DptrIterator> u1(V.data+block_len-1, V.data+V.size, block_len);

    if (axis == 1) {
        if (has_low_dirichlet) {
            thrust::copy(
                low_dirichlet.data,
                low_dirichlet.data + low_dirichlet.size,
                u1.begin()
                );
        }
        if (has_high_dirichlet) {
            thrust::copy(
                high_dirichlet.data,
                high_dirichlet.data + high_dirichlet.size,
                u0.begin()
                );
        }
    } else {
        if (has_low_dirichlet) {
            thrust::copy(low_dirichlet.data,
                    low_dirichlet.data + low_dirichlet.size,
                    V.data);
        }
        if (has_high_dirichlet) {
            thrust::copy(high_dirichlet.data,
                    high_dirichlet.data + high_dirichlet.size,
                    V.data + V.size - V.shape[1]);
        }
        V.transpose(1);
    }


    // Upper and lower edge cases simultaneously
    thrust::for_each(
        make_zip_iterator(make_tuple(V.tempspace, mid, V.data, sup, V.data+1,
                V.tempspace+N-1, sub+N-1, V.data+N-2, mid+N-1, V.data+N-1)
            ),
        make_zip_iterator(make_tuple(V.tempspace+1, mid+1, V.data+1, sup+1,
                V.data+2, V.tempspace+N, sub+N, V.data+N-1, mid+N, V.data+N)
            ),
        zipdotTopAndBottom()
    );

    // Regular dot product
    thrust::transform(
        make_zip_iterator(make_tuple(sub+1, mid+1, sup+1)),
        make_zip_iterator(make_tuple(sub+N-1, mid+N-1, sup+N-1)),
        make_zip_iterator(make_tuple(V.data, V.data+1, V.data+2)),
        V.tempspace+1,
        zipdot3()
    );


    if (is_folded()) {
        std::swap(V.tempspace, V.data);
        fold_vector(V, true);
        std::swap(V.tempspace, V.data);
    }


    if (has_residual) {
        thrust::transform(
                V.tempspace,
                V.tempspace + V.size,
                R.data,
                V.data,
                thrust::plus<double>());
    } else {
        if (V.owner) {
            std::swap(V.tempspace, V.data);
        } else {
            thrust::copy(V.tempspace, V.tempspace + V.size, V.data);
        }
    }


    if (axis == 0) {
        V.transpose(1);
    }
    FULLTRACE;
    return;
}


struct periodic_from_to_mask : thrust::unary_function<int, bool> {
    int begin;
    int end;
    int period;

    periodic_from_to_mask(int begin, int end, int period)
        : begin(begin-1), end(end+1), period(period) {}

    __host__ __device__
    bool operator()(int idx) {
        return (idx % period != begin && idx % period != end);
    }
};

/*
* Add a second BandedOperator to this one.
* Does not alter self.R, the residual vector.
*/
void _TriBandedOperator::add_operator(_TriBandedOperator &other) {
    FULLTRACE;
    if (is_folded() || other.is_folded()) {
        DIE("Cannot add folded (diagonalized) operators");
    }
    int begin = has_low_dirichlet;
    int end = block_len-1 - has_high_dirichlet;
    int o, to, fro;
    for (int i = 0; i < 3; ++i) {
        fro = to = i;
        o = 1-i;
        if (o == 0) {
            thrust::transform_if(
                    diags.data + diags.idx(to, 0),
                    diags.data + diags.idx(to, 0) + operator_rows,
                    other.diags.data + diags.idx(fro, 0),
                    make_counting_iterator(0),
                    diags.data + diags.idx(to, 0),
                    thrust::plus<double>(),
                    periodic_from_to_mask(begin, end, block_len));
        } else {
            thrust::transform(
                    other.diags.data + diags.idx(fro, 0),
                    other.diags.data + diags.idx(fro, 0) + other.diags.shape[1],
                    diags.data + diags.idx(to, 0),
                    diags.data + diags.idx(to, 0),
                    thrust::plus<double>());
        }
    }

    if (other.top_fold_status == CAN_FOLD) {
        int them = other.top_factors.size;
        int us = top_factors.size;
        if (them != us) {
            DIE("Top_factors are different sizes:" << us << ", " << them);
        }
        thrust::transform(
            top_factors.data,
            top_factors.data + top_factors.size,
            other.top_factors.data,
            top_factors.data,
            thrust::plus<double>());
    }

    if (other.bottom_fold_status == CAN_FOLD) {
        int them = other.bottom_factors.size;
        int us = bottom_factors.size;
        if (them != us) {
            DIE("Bottom_factors are different sizes:" << us << ", " << them);
        }
        thrust::transform(
            bottom_factors.data,
            bottom_factors.data + bottom_factors.size,
            other.bottom_factors.data,
            bottom_factors.data,
            thrust::plus<double>());
    }

    thrust::transform(
            R.data,
            R.data + R.size,
            other.R.data,
            R.data,
            thrust::plus<double>());
    FULLTRACE;
}


/* Add a scalar to the main diagonal.
* Does not alter the residual vector.
*/
void _TriBandedOperator::add_scalar(double val) {
    FULLTRACE;

    int begin = has_low_dirichlet;
    int end = block_len-1 - has_high_dirichlet;

    thrust::transform_if(
            diags.data + diags.idx(main_diag, 0),
            diags.data + diags.idx(main_diag, 0) + operator_rows,
            make_constant_iterator(val),
            make_counting_iterator(0),
            diags.data + diags.idx(main_diag, 0),
            thrust::plus<double>(),
            periodic_from_to_mask(begin, end, block_len));
    FULLTRACE;
}


bool _TriBandedOperator::is_folded() {
    return (top_fold_status == FOLDED || bottom_fold_status == FOLDED);
}


void _TriBandedOperator::solve(SizedArray<double> &V) {
    FULLTRACE;
    if (top_fold_status == CAN_FOLD || bottom_fold_status == CAN_FOLD) {
        DIE("Must be tridiagonal to apply inverse operator on GPU.");
    }
    const unsigned N = V.size;

    if (has_low_dirichlet) {
        thrust::copy(low_dirichlet.data,
                low_dirichlet.data + low_dirichlet.size,
                V.data);
    }
    if (has_high_dirichlet) {
        thrust::copy(high_dirichlet.data,
                high_dirichlet.data + high_dirichlet.size,
                V.data + V.size - V.shape[1]);
    }

    if (axis == 0) {
        V.transpose(1);
    }

    if (has_residual) {
        thrust::transform(V.data, V.data + V.size,
                R.data,
                V.data,
                thrust::minus<double>());
    }

    if (is_folded()) {
        fold_vector(V);
    }

    status = cusparseDgtsvStridedBatch(handle, N,
                sub.get(), mid.get(), sup.get(),
                V.data.get(),
                1, N);

    /*
     * if (block_len > 256) {
     *     DIE("Block_len Too big!");
     * }
     * if (blocks * block_len > V.size) {
     *     DIE("Indexing is wrong. Too large.");
     * }
     * triDiagonalSystemSolve<<<blocks, block_len>>>(V.size, sub.get(), mid.get(), sup.get(), V.data.get());
     * hipDeviceSynchronize();
     */

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        DIE("CUSPARSE tridiag system solve failed.");
    }

    if (axis == 0) {
        V.transpose(1);
    }
    FULLTRACE;
    return;
}


template <typename Tuple, typename OP>
struct curry :
    public thrust::unary_function<Tuple, typename OP::result_type> {

    OP f;

    __host__ __device__
    typename OP::result_type operator()(Tuple t) {
        using thrust::get;
        return  f(get<0>(t), get<1>(t));
    }
};

template <typename Tuple, typename Result>
struct add_multiply3 : public thrust::unary_function<Tuple, Result> {
    Result direction;
    add_multiply3(Result x) : direction(x) {}
    __host__ __device__
    Result operator()(Tuple t) {
        using thrust::get;
        return  get<0>(t) + direction * get<1>(t) * get<2>(t);
    }
};


void _TriBandedOperator::fold_vector(SizedArray<double> &vector, bool unfold) {
    FULLTRACE;

    typedef thrust::tuple<REAL_t,REAL_t,REAL_t> REALTuple;

    strided_range<DptrIterator> u0(vector.data, vector.data + vector.size, block_len);
    strided_range<DptrIterator> u1(vector.data+1, vector.data + vector.size, block_len);

    strided_range<DptrIterator> un(vector.data+block_len-1, vector.data + vector.size, block_len);
    strided_range<DptrIterator> un1(vector.data+block_len-2, vector.data + vector.size, block_len);

    // Top fold
    if (top_fold_status == FOLDED) {
        /* LOG("Folding top. direction("<<unfold<<") top_factors("<<top_factors<<")"); */
        thrust::transform(
            make_zip_iterator(make_tuple(u0.begin(), u1.begin(), top_factors.data)),
            make_zip_iterator(make_tuple(u0.end(), u1.end(), top_factors.data + top_factors.size)),
            u0.begin(),
            add_multiply3<REALTuple, REAL_t>(unfold ? -1 : 1));
    }

    if (bottom_fold_status == FOLDED) {
        /* LOG("Folding bottom. direction("<<unfold<<") bottom_factors("<<bottom_factors<<")"); */
        thrust::transform(
            make_zip_iterator(make_tuple(un.begin(), un1.begin(), bottom_factors.data)),
            make_zip_iterator(make_tuple(un.end(), un1.end(), bottom_factors.data + bottom_factors.size)),
            un.begin(),
            add_multiply3<REALTuple, REAL_t>(unfold ? -1 : 1));
    }

    FULLTRACE;
}


void _TriBandedOperator::diagonalize() {
    FULLTRACE;
    /* LOG("Before folding: " << diags); */
    if (bottom_fold_status == CAN_FOLD) {
        /* LOG("Bottom:" << bottom_fold_status); */
        fold_bottom();
        /* LOG("Bottom:" << bottom_fold_status); */
    }
    if (top_fold_status == CAN_FOLD) {
        /* LOG("Top:" << top_fold_status); */
        fold_top();
        /* LOG("Top:" << top_fold_status); */
    }
    /* LOG("After folding: " << diags); */
    FULLTRACE;
}


void _TriBandedOperator::undiagonalize() {
    FULLTRACE;
    if (bottom_fold_status == FOLDED) {
        /* LOG("Bottom:" << bottom_fold_status); */
        fold_bottom(true);
        /* LOG("Bottom:" << bottom_fold_status); */
    }
    if (top_fold_status == FOLDED) {
        /* LOG("Top:" << top_fold_status); */
        fold_top(true);
        /* LOG("Top:" << top_fold_status); */
    }
    FULLTRACE;
}


/* These fold the third element of first row and third from last element of last
 * row into the neighboring row, resulting in a tridiagonal system.
 */
template <typename Tuple>
struct fold_operator : public thrust::unary_function<Tuple, void> {
    __host__ __device__
    void operator()(Tuple t) {
        using thrust::get;
        get<5>(t) = get<1>(t) == 0 ? 0 : -get<5>(t) / get<1>(t);
        get<0>(t) += get<3>(t) * get<5>(t);
        get<2>(t) += get<4>(t) * get<5>(t);
    }
};
template <typename Tuple>
struct unfold_operator : public thrust::unary_function<Tuple, void> {
    __host__ __device__
    void operator()(Tuple t) {
        using thrust::get;
        get<0>(t) -= get<3>(t) * get<5>(t);
        get<2>(t) -= get<4>(t) * get<5>(t);
        get<5>(t) *= -get<1>(t);
    }
};

void _TriBandedOperator::fold_top(bool unfold) {
    FULLTRACE;
    typedef thrust::tuple<REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&> REALTuple;
    typedef thrust::device_ptr<REAL_t> Ptr;

    strided_range<Ptr> c0 (sup  , sup+operator_rows, block_len);
    strided_range<Ptr> c1 (sup+1, sup+operator_rows, block_len);
    strided_range<Ptr> b0 (mid  , mid+operator_rows, block_len);
    strided_range<Ptr> b1 (mid+1, mid+operator_rows, block_len);
    strided_range<Ptr> a1 (sub+1, sub+operator_rows, block_len);

    if (unfold) {
        thrust::for_each(
            make_zip_iterator(
                make_tuple(
                    c0.begin(), c1.begin(),
                    b0.begin(), b1.begin(),
                                a1.begin(),
                    top_factors.data
                )
            ),
            make_zip_iterator(
                make_tuple(
                    c0.end(), c1.end(),
                    b0.end(), b1.end(),
                            a1.end(),
                    top_factors.data + top_factors.size
                )
            ),
            unfold_operator <REALTuple>()
        );
    } else {
        thrust::for_each(
            make_zip_iterator(
                make_tuple(
                    c0.begin(), c1.begin(),
                    b0.begin(), b1.begin(),
                                a1.begin(),
                    top_factors.data
                )
            ),
            make_zip_iterator(
                make_tuple(
                    c0.end(), c1.end(),
                    b0.end(), b1.end(),
                            a1.end(),
                    top_factors.data + top_factors.size
                )
            ),
            fold_operator<REALTuple>()
        );
    }
    if (unfold) top_fold_status = CAN_FOLD;
    else top_fold_status = FOLDED;
    FULLTRACE;
}


void _TriBandedOperator::fold_bottom(bool unfold) {
    FULLTRACE;
    typedef thrust::tuple<REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&, REAL_t&> REALTuple;
    typedef thrust::device_ptr<REAL_t> Ptr;

    strided_range<Ptr> cn1(sup+(block_len-2)  , sup+operator_rows, block_len);
    strided_range<Ptr> bn (mid+(block_len-1)  , mid+operator_rows, block_len);
    strided_range<Ptr> bn1(mid+(block_len-1)-1, mid+operator_rows, block_len);
    strided_range<Ptr> an (sub+(block_len-1), sub+operator_rows, block_len);
    strided_range<Ptr> an1(sub+(block_len-1)-1, sub+operator_rows, block_len);

    if (unfold) {
        thrust::for_each(
            make_zip_iterator(
                make_tuple(
                    an.begin(), an1.begin(),
                    bn.begin(), bn1.begin(),
                                cn1.begin(),
                    bottom_factors.data
                )
            ),
            make_zip_iterator(
                make_tuple(
                    an.end(), an1.end(),
                    bn.end(), bn1.end(),
                            cn1.end(),
                    bottom_factors.data + bottom_factors.size
                )
            ),
            unfold_operator<REALTuple>()
        );
    } else {
        thrust::for_each(
            make_zip_iterator(
                make_tuple(
                    an.begin(), an1.begin(),
                    bn.begin(), bn1.begin(),
                                cn1.begin(),
                    bottom_factors.data
                )
            ),
            make_zip_iterator(
                make_tuple(
                    an.end(), an1.end(),
                    bn.end(), bn1.end(),
                            cn1.end(),
                    bottom_factors.data + bottom_factors.size
                )
            ),
            fold_operator<REALTuple>()
        );
    }

    if (unfold) bottom_fold_status = CAN_FOLD;
    else bottom_fold_status = FOLDED;
    FULLTRACE;
}


void _TriBandedOperator::vectorized_scale(SizedArray<double> &vector) {
    FULLTRACE;
    Py_ssize_t vsize = vector.size;
    Py_ssize_t block_len = operator_rows / blocks;

    tiled_range<DptrIterator> v(vector.data, vector.data + vector.size, block_len);
    typedef tiled_range<DptrIterator>::iterator TiledIterator;

    strided_range<TiledIterator> u0(v.begin(), v.end(), block_len);
    strided_range<TiledIterator> u1(v.begin()+block_len-1, v.end(), block_len);

    /*
     * LOG("op_rows("<<operator_rows<<") vsize("<<vsize<<") "
     *     "v.d.size("<<vector.size<<") "
     *     "v.size()("<<v.end()-v.begin()<<") "
     *     "diags.shape("<<diags.shape[0]<<","<<diags.shape[1]<<") "
     *     "diags.idx(1,0)("<<diags.idx(1,0)<<") "
     *     );
     * LOG("diags.name("<<diags.name<<")");
     * LOG("diags.idx(0,op)("<<diags.idx(0,0)+operator_rows<<")");
     */

    if (is_folded()) {
        DIE("Cannot scale diagonalized operator.");
    }

    if (operator_rows % vsize != 0) {
        DIE("Vector length does not divide "
            "evenly into operator size. Cannot scale."
            << "\n vsize("<<vsize<<") operator_rows("<<operator_rows<<")");
    }

    if (vsize == 0) {
        DIE("vsize == 0");
    }

    for (Py_ssize_t row = 0; row < 3; ++row) {
        int o = 1 - row;
        if (o > 0) { // upper diags
            thrust::transform(
                    diags.data + diags.idx(row, 0),
                    diags.data + diags.idx(row, 0) + operator_rows - o,
                    v.begin(),
                    diags.data + diags.idx(row, 0),
                    thrust::multiplies<REAL_t>());
        } else if (o < 0) { // lower diags
            thrust::transform(
                    diags.data + diags.idx(row, -o),
                    diags.data + diags.idx(row, 0) + operator_rows,
                    v.begin() + -o,
                    diags.data + diags.idx(row, -o),
                    thrust::multiplies<REAL_t>());
        } else { // main diagonal
            int begin = has_low_dirichlet;
            int end = block_len-1 - has_high_dirichlet;
            thrust::transform_if(
                    diags.data + diags.idx(row, 0),
                    diags.data + diags.idx(row, 0) + operator_rows,
                    v.begin(),
                    make_counting_iterator(0),
                    diags.data + diags.idx(row, 0),
                    thrust::multiplies<REAL_t>(),
                    periodic_from_to_mask(begin, end, block_len));
        }

    }

    /* We check dirichlet to avoid multiplying by 1 */
    if (top_fold_status == CAN_FOLD && !has_low_dirichlet) {
        thrust::transform(
            top_factors.data,
            top_factors.data+top_factors.size,
            u0.begin(),
            top_factors.data,
            thrust::multiplies<REAL_t>());
    }
    if (bottom_fold_status == CAN_FOLD && !has_high_dirichlet) {
        thrust::transform(
            bottom_factors.data,
            bottom_factors.data+bottom_factors.size,
            u1.begin(),
            bottom_factors.data,
            thrust::multiplies<REAL_t>());
    }

    /* LOG("Scaled data."); */
    thrust::transform(R.data, R.data + R.size,
            v.begin(),
            R.data,
            thrust::multiplies<REAL_t>());
    /* LOG("Scaled R."); */
    FULLTRACE;
    return;
}

_TriBandedOperator *for_vector(SizedArray<double> &V, Py_ssize_t blocks,
        Py_ssize_t derivative, Py_ssize_t axis) {

    int blksz = V.size;
    int operator_rows = blocks * blksz;

    SizedArray<double> data(operator_rows * 3, "data");
    SizedArray<double> R(operator_rows, "R");
    SizedArray<double> high_dirichlet(blocks, "high_dirichlet");
    SizedArray<double> low_dirichlet(blocks, "low_dirichlet");
    SizedArray<double> top_factors(blocks, "top_factors");
    SizedArray<double> bottom_factors(blocks, "bottom_factors");
    SizedArray<double> deltas(blksz, "deltas");

    data.shape[0] = 3;
    data.shape[1] = operator_rows;
    data.ndim = 2;
    data.sanity_check();

    Dptr sup, mid, sub;
    sup = data.data;
    mid = sup + operator_rows;
    sub = mid + operator_rows;

    thrust::fill(R.data, R.data + R.size, 0);
    thrust::adjacent_difference(V.data, V.data + V.size, deltas.data);
    // XXX
    deltas.data[0] = NaN;
    tiled_range<DptrIterator> delta_rep(deltas.data, deltas.data + deltas.size, blocks);

    if (derivative == 1) {
        if (axis == 0) {
            spot_first(sup, mid, sub, delta_rep.begin(),
                    low_dirichlet.data, R.data, operator_rows, blksz);
        } else if (axis == 1) {
            var_first(sup, mid, sub, delta_rep.begin(),
                    operator_rows, blksz);
        } else {
            throw std::invalid_argument("axis must be one of (0, 1)");
        }
    } else if (derivative == 2) {
        if (axis == 0) {
            spot_second(sup, mid, sub, delta_rep.begin(),
                low_dirichlet.data, R.data, operator_rows, blksz);
        } else if (axis == 1) {
            var_second(sup, mid, sub, delta_rep.begin(),
                R.data, bottom_factors.data, operator_rows, blksz);
        } else {
            throw std::invalid_argument("derivative must be one of (1, 2)");
        }
    }

    bool has_high_dirichlet = false;
    bool has_low_dirichlet = axis == 0;
    std::string top_fold_status = "CANNOT_FOLD";
    std::string bottom_fold_status = axis == 1 && derivative == 2 ? "CAN_FOLD" : "CANNOT_FOLD";
    bool has_residual = axis == 0;

    return new _TriBandedOperator(
            data, R, high_dirichlet, low_dirichlet, top_factors, bottom_factors,
            axis, operator_rows, blocks, has_high_dirichlet, has_low_dirichlet,
            top_fold_status, bottom_fold_status, has_residual);
}
