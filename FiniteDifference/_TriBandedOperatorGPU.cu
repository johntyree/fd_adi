#include "hip/hip_runtime.h"

#include "GNUC_47_compat.h"

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/version.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include "tiled_range.h"
#include "strided_range.h"

#include <algorithm>
#include <cstdlib>

#include <iostream>
#include <vector>
#include <iterator>
#include <cassert>
#include <stdexcept>

#include <sys/select.h>

#include <hipsparse.h>

#include "_TriBandedOperatorGPU.cuh"

template <typename T, typename U>
int find_index(T haystack, U needle, int max) {
    FULLTRACE;
    int idx;
    for (idx = 0; idx < max; ++idx) {
        if (haystack[idx] == needle) break;
    }
    if (idx >= max) {
        /* LOG("Did not find "<<needle<<" before reaching index "<<max<<"."); */
        /* std::cout << '\t'; */
        /* std::cout << "Haystack [ "; */
        /* for (idx = 0; idx < max; ++idx) { */
            /* std::cout << haystack[idx] << " "; */
        /* } */
        /* std::cout << "]"; ENDL; */
        idx = -1;
    }
    FULLTRACE;
    return idx;
}

_TriBandedOperator::_TriBandedOperator(
        SizedArray<double> &data,
        SizedArray<double> &R,
        SizedArray<int> &offsets,
        SizedArray<double> &high_dirichlet,
        SizedArray<double> &low_dirichlet,
        SizedArray<double> &top_factors,
        SizedArray<double> &bottom_factors,
        unsigned int axis,
        Py_ssize_t operator_rows,
        Py_ssize_t blocks,
        bool has_high_dirichlet,
        bool has_low_dirichlet,
        bool has_top_factors,
        bool has_bottom_factors,
        bool has_residual
        ) :
    diags(data),
    R(R),
    offsets(offsets),
    high_dirichlet(high_dirichlet),
    low_dirichlet(low_dirichlet),
    top_factors(top_factors),
    bottom_factors(bottom_factors),
    axis(axis),
    main_diag(find_index(offsets.data, 0, offsets.size)),
    operator_rows(operator_rows),
    blocks(blocks),
    block_len(operator_rows / blocks),
    sup(diags.data.ptr()),
    mid(diags.data.ptr() + operator_rows),
    sub(diags.data.ptr() + 2*operator_rows),
    has_high_dirichlet(has_high_dirichlet),
    has_low_dirichlet(has_low_dirichlet),
    has_top_factors(has_top_factors),
    has_bottom_factors(has_bottom_factors),
    has_residual(has_residual),
    is_tridiagonal(offsets.size == 3 && main_diag != -1)
    {
        verify_diag_ptrs();
        status = hipsparseCreate(&handle);
        if (status != HIPSPARSE_STATUS_SUCCESS) {
            DIE("CUSPARSE Library initialization failed.");
        }
    }

void _TriBandedOperator::verify_diag_ptrs() {
    FULLTRACE;
    if (sup.get() == 0 || mid.get() == 0 || sub.get() == 0) {
        DIE("Diag pointers aren't non-null");
    }
    if (main_diag == -1) {
        /* LOG("No main diag means not tridiagonal, hopefully."); */
        return;
    }
    if (offsets.size != 3) {
        /* LOG("Not tridiagonal. Skipping diag ptrs check."); */
        return;
    }
    int idx;
    /* LOG("main_diag("<<main_diag<<")"); */
    idx = diags.idx(main_diag-1, 0);
    if (*sup != diags.data[idx]
            || (sup.get() != (&diags.data[diags.idx(0,0)]).get())) {
        DIE("sup[0] = " << *sup << " <->  " << diags.get(0,0)
            << "\n\tsup = " << sup.get() << " <->  "
                << (&diags.data[diags.idx(0, 0)]).get());
    }
    if (*mid != diags.data[diags.idx(main_diag, 0)]
            || (mid.get() != (&diags.data[diags.idx(main_diag, 0)]).get())) {
        DIE("mid[0] = " << *mid << " !=  " << diags.get(main_diag,0)
            << "\n\tmid = " << mid.get() << " <->  "
                << (&diags.data[diags.idx(main_diag, 0)]).get());
    }
    if (*sub != diags.data[diags.idx(main_diag+1, 0)]
            || (sub.get() != (&diags.data[diags.idx(main_diag+1, 0)]).get())) {
        DIE("sub[0] = " << *sub << " !=  " << diags.get(main_diag+1,0)
            << "\n\tsub = " << sub.get() << " <->  "
                << (&diags.data[diags.idx(main_diag+1, 0)]).get());
    }
    FULLTRACE;
}


struct zipdot3 : thrust::binary_function<const Triple &, const Triple &, REAL_t> {
    __host__ __device__
    REAL_t operator()(const Triple &diags, const Triple &x) {
        using thrust::get;
        const REAL_t a = get<0>(diags);
        const REAL_t b = get<1>(diags);
        const REAL_t c = get<2>(diags);
        const REAL_t x0 = get<0>(x);
        const REAL_t x1 = get<1>(x);
        const REAL_t x2 = get<2>(x);
        return a*x0 + b*x1 + c*x2;
    }
};

SizedArray<double> *_TriBandedOperator::apply(SizedArray<double> &V) {
    FULLTRACE;
    using std::cout;
    using std::endl;
    const unsigned N = V.size;
    GPUVec<REAL_t> out(N);

    GPUVec<REAL_t> &in = V.data;

    if (!is_tridiagonal) {
        DIE("Can only apply tridiagonal operators when on the GPU.");
    }


    if (has_low_dirichlet) {
        /* print "Setting V[0,:] to", self.dirichlet[0] */
        // Some kind of thrust thing?
        /* V[...,0] = low_dirichlet[i] */
        thrust::copy(low_dirichlet.data.begin(),
                low_dirichlet.data.end(),
                in.begin());
    }
    if (has_high_dirichlet) {
        /* print "Setting V[0,:] to", self.dirichlet[0] */
        // Some kind of thrust thing?
        /* V[...,-1] = high_dirichlet[i] */
        thrust::copy(high_dirichlet.data.begin(),
                high_dirichlet.data.end(),
                in.end() - V.shape[1]);
    }

    if (axis == 0) {
        V.transpose(1);
    }



    GPUVec<REAL_t> a(sub, sub+N);
    GPUVec<REAL_t> b(mid, mid+N);
    GPUVec<REAL_t> c(sup, sup+N);

    out[0] = b[0]*in[0] + c[0]*in[1];
    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(a.begin()+1, b.begin()+1, c.begin()+1)),
        thrust::make_zip_iterator(thrust::make_tuple(a.end()-1, b.end()-1, c.end()-1)),
        thrust::make_zip_iterator(thrust::make_tuple(in.begin(), in.begin()+1, in.begin()+2)),
        out.begin()+1,
        zipdot3()
    );
    out[N-1] = a[N-1]*in[N-2] + b[N-1]*in[N-1];

    SizedArray<double> *U = new SizedArray<double>(out,
            V.ndim, V.shape, "CPP Solve U from V");

    /* if (has_residual) { */
        /* thrust::transform(U->data.begin(), U->data.end(), */
                /* R.data.begin(), */
                /* U->data.begin(), */
                /* thrust::plus<double>()); */
    /* } */

    /* ret = ret.reshape(V.shape) */

    /* t = range(V.ndim) */
    /* utils.rolllist(t, V.ndim-1, self.axis) */

    /* if (axis == 0) { */
        /* U.transpose(); */
    /* } */
    /* return ret; */
    FULLTRACE;
    return U;
}


struct periodic_from_to_mask : thrust::unary_function<int, bool> {
    int begin;
    int end;
    int period;

    periodic_from_to_mask(int begin, int end, int period)
        : begin(begin-1), end(end+1), period(period) {
        }

    __host__ __device__
    bool operator()(int idx) {
        return (idx % period != begin && idx % period != end);
    }
};

void _TriBandedOperator::add_operator(_TriBandedOperator &other) {
    /* LOG("Adding operator @ " << &other << " to this one @ " << this); */
    /*
    * Add a second BandedOperator to this one.
    * Does not alter self.R, the residual vector.
    */
    FULLTRACE;
    int begin = has_low_dirichlet;
    int end = block_len-1 - has_high_dirichlet;
    int o, to, fro;
    for (int i = 0; i < other.offsets.size; i++) {
        fro = i;
        o = other.offsets.get(i);
        to = find_index(offsets.data, o, offsets.size);
        if (offsets.get(to) != o) {
            std::cout << std::endl;
            std::cout << "to: " << to << "(";
            /* print_array(&offsets(0), offsets.size); */
            std::cout << offsets.data;
            std::cout << ")";
            std::cout << "fro: " << fro << "(";
            std::cout << other.offsets.data;
            /* print_array(&other.offsets(0), other.offsets.size); */
            std::cout << ")" << std::endl;
            assert(offsets.get(to) == o);
        }
        /* LOG("Adding offset " << o << "."); */
        if (o == 0) {
            thrust::transform_if(
                    &diags.data[diags.idx(to, 0)],
                    &diags.data[diags.idx(to, 0)] + operator_rows,
                    &other.diags.data[diags.idx(fro, 0)],
                    thrust::make_counting_iterator(0),
                    &diags.data[diags.idx(to, 0)],
                    thrust::plus<double>(),
                    periodic_from_to_mask(begin, end, block_len));
        } else {
            thrust::transform(
                    &other.diags.data[diags.idx(fro, 0)],
                    &other.diags.data[diags.idx(fro, 0)] + other.diags.shape[1],
                    &diags.data[diags.idx(to, 0)],
                    &diags.data[diags.idx(to, 0)],
                    thrust::plus<double>());
        }
    }
    /* LOG("Adding R."); */
    thrust::transform(
            R.data.begin(),
            R.data.end(),
            other.R.data.begin(),
            R.data.begin(),
            thrust::plus<double>());
    FULLTRACE;
}



void _TriBandedOperator::add_scalar(double val) {
    FULLTRACE;
    /* Add a scalar to the main diagonal.
     * Does not alter the residual vector.
     */
    // We add it to the main diagonal.

    int begin = has_low_dirichlet;
    int end = block_len-1 - has_high_dirichlet;

    /* LOG("has_low("<<has_low_dirichlet<<") " */
        /* "has_high("<<has_high_dirichlet<<") " */
        /* "blocklen("<<block_len<<") "); */

    thrust::transform_if(
            &diags.data[diags.idx(main_diag, 0)],
            &diags.data[diags.idx(main_diag, 0)] + operator_rows,
            thrust::make_constant_iterator(val),
            thrust::make_counting_iterator(0),
            &diags.data[diags.idx(main_diag, 0)],
            thrust::plus<double>(),
            periodic_from_to_mask(begin, end, block_len));
    FULLTRACE;
}

bool _TriBandedOperator::is_folded() {
    return has_top_factors || has_bottom_factors;
}



int _TriBandedOperator::solve(SizedArray<double> &V) {
    FULLTRACE;
    if (!is_tridiagonal) {
        DIE("Can only solve tridiagonal systems when on the GPU.");
    }
    verify_diag_ptrs();

    /* std::cout << "Begin C Solve\n"; */
    /* std::cout << "Copy Host->Dev... " << V.data << ' '; */
    GPUVec<double> d_V(V.data);
    GPUVec<double> d_sup(sup, sup+V.size);
    GPUVec<double> d_mid(mid, mid+V.size);
    GPUVec<double> d_sub(sub, sub+V.size);
    /* std::cout << "OK\n"; */

    /* std::cout << "CUSPARSE... "; */
    status = cusparseDgtsvStridedBatch(handle, V.size,
            d_sub.raw(), d_mid.raw(), d_sup.raw(),
            d_V.raw(),
            1, V.size);
    hipDeviceSynchronize();
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << "CUSPARSE tridiag system solve failed." << std::endl;
        return 1;
    }
    /* std::cout << "OK\n"; */

    /* std::cout << "Copy Dev->Host... " << d_V << ' '; */
    thrust::copy(d_V.begin(), d_V.end(), V.data.begin());
    /* std::cout << "OK\n"; */
    /* std::cout << "End C Solve\n"; */
    FULLTRACE;
    return 0;
}

/*
 * cpdef fold_vector(self, double[:] v, unfold=False):
 *     cdef int direction, u0, u1, un ,un1
 *     blocks = self.blocks
 *     block_len = self.shape[0] // blocks
 *
 *     for b in range(blocks):
 *         u0 = b*block_len
 *         u1 = u0 + 1
 *         un = (b+1)*block_len - 1
 *         un1 = un - 1
 *         # print u0, u1, un1, un
 *         # print "[%f, %f .. %f, %f]" % (v[u0], v[u1], v[un1], v[un])
 *         direction = -1 if unfold else 1
 *         if self.top_factors is not None:
 *             v[u0] += direction * v[u1]  * self.top_factors[b]
 *         if self.bottom_factors is not None:
 *             v[un] += direction * v[un1] * self.bottom_factors[b]
 *     return np.asarray(v)
 */

template <typename Tuple, typename OP>
struct curry : public thrust::unary_function<Tuple, typename OP::result_type> {

    OP f;

    __host__ __device__
    typename OP::result_type operator()(Tuple t) {
        using thrust::get;
        return  f(get<0>(t), get<1>(t));
    }
};

template <typename Tuple, typename Result>
struct fold_it : public thrust::unary_function<Tuple, Result> {

    Result direction;

    fold_it(Result x) : direction(x) {}

    __host__ __device__
    Result operator()(Tuple t) {
        using thrust::get;
        return  get<0>(t) + direction * get<1>(t) * get<2>(t);
    }
};


void _TriBandedOperator::fold_vector(GPUVec<double> &vector, bool unfold) {
    FULLTRACE;

    typedef thrust::device_vector<REAL_t>::iterator Iterator;

    strided_range<Iterator> u0(vector.begin(), vector.end(), block_len);
    strided_range<Iterator> u1(vector.begin()+1, vector.end(), block_len);

    strided_range<Iterator> un(vector.begin()+block_len-1, vector.end(), block_len);
    strided_range<Iterator> un1(vector.begin()+block_len-2, vector.end(), block_len);

    using thrust::make_zip_iterator;
    using thrust::make_tuple;
    using thrust::get;

    typedef thrust::tuple<REAL_t,REAL_t,REAL_t> REALTuple;
    typedef thrust::device_vector<REAL_t>::iterator Iterator;

    LOG("has_top_factors("<<has_top_factors<<") has_bottom_factors("<<has_bottom_factors<<")");

    // Top fold
    if (has_top_factors) {
        /* LOG("Folding top. direction("<<unfold<<") top_factors("<<top_factors<<")"); */
        thrust::transform(
            make_zip_iterator(make_tuple(u0.begin(), u1.begin(), top_factors.data.begin())),
            make_zip_iterator(make_tuple(u0.end(), u1.end(), top_factors.data.end())),
            u0.begin(),
            fold_it<REALTuple, REAL_t>(unfold ? -1 : 1));
    }

    if (has_bottom_factors) {
        /* LOG("Folding bottom. direction("<<unfold<<") bottom_factors("<<bottom_factors<<")"); */
        thrust::transform(
            make_zip_iterator(make_tuple(un.begin(), un1.begin(), bottom_factors.data.begin())),
            make_zip_iterator(make_tuple(un.end(), un1.end(), bottom_factors.data.end())),
            un.begin(),
            fold_it<REALTuple, REAL_t>(unfold ? -1 : 1));
    }

    FULLTRACE;
}



void _TriBandedOperator::vectorized_scale(SizedArray<double> &vector) {
    FULLTRACE;
    Py_ssize_t vsize = vector.size;
    Py_ssize_t noffsets = offsets.size;
    Py_ssize_t block_len = operator_rows / blocks;

    typedef thrust::device_vector<REAL_t>::iterator Iterator;
    tiled_range<Iterator> v(vector.data.begin(), vector.data.end(), operator_rows / vsize);
    /*
     * LOG("op_rows("<<operator_rows<<") vsize("<<vsize<<") "
     *     "v.d.size("<<vector.data.size()<<") "
     *     "v.size()("<<v.end()-v.begin()<<") "
     *     "diags.shape("<<diags.shape[0]<<","<<diags.shape[1]<<") "
     *     "diags.idx(1,0)("<<diags.idx(1,0)<<") "
     *     );
     * LOG("diags.name("<<diags.name<<")");
     * LOG("diags.idx(0,op)("<<diags.idx(0,0)+operator_rows<<")");
     */

    if (operator_rows % vsize != 0) {
        DIE("Vector length does not divide "
            "evenly into operator size. Cannot scale."
            << "\n vsize("<<vsize<<") operator_rows("<<operator_rows<<")");
    }
    if ((size_t)vsize != vector.data.size()) {DIE("vsize != vector.data.size()")}
    if (vsize == 0) {DIE("vsize == 0")}

    if (has_low_dirichlet) {
        for (Py_ssize_t b = 0; b < blocks; ++b) {
            vector.data[vector.idx(b*block_len % vsize)] = 1;
        }
    }

    if (has_high_dirichlet) {
        for (Py_ssize_t b = 0; b < blocks; ++b) {
            vector.data[vector.idx((b+1)*block_len - 1 % vsize)] = 1;
        }
    }

    for (Py_ssize_t row = 0; row < noffsets; ++row) {
        int o = offsets.get(row);
        if (o >= 0) { // upper diags
            thrust::transform(diags.data.begin() + diags.idx(row, 0),
                    diags.data.begin() + diags.idx(row, 0) + operator_rows - o,
                    v.begin(),
                    diags.data.begin() + diags.idx(row, 0),
                    thrust::multiplies<REAL_t>());
        } else { // lower diags
            thrust::transform(diags.data.begin() + diags.idx(row, -o),
                    diags.data.begin() + diags.idx(row, 0) + operator_rows,
                    v.begin() + -o,
                    diags.data.begin() + diags.idx(row, -o),
                    thrust::multiplies<REAL_t>());
        }
    }
    /* LOG("Scaled data."); */
    thrust::transform(R.data.begin(), R.data.end(),
            v.begin(),
            R.data.begin(),
            thrust::multiplies<REAL_t>());
    /* LOG("Scaled R."); */
    FULLTRACE;
    return;
}

int main () {

    thrust::host_vector<double> a(10);
    int block_len = 5;
    int begin = 1;
    int end = block_len-1 - 1;

    thrust::transform_if(
            a.begin(),
            a.end(),
            thrust::make_constant_iterator(2),
            thrust::make_counting_iterator(0),
            a.begin(),
            thrust::plus<double>(),
            periodic_from_to_mask(begin, end, block_len));

    printf("\n");
    print_array(a.data(), a.size());
    return 0;
}
